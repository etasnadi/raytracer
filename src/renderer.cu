#include "hip/hip_runtime.h"
#include "renderer.h"

#include <cmath>
#include <cstdint>
#include <iostream>
#include <limits>
#include <memory>
#include <sstream>
#include <vector>

#include "hip/hip_runtime.h"

#include "basic_types.h"
#include "cudastuff.h"
#include "math.h"
#include "raytracer_basics.h"
#include "room_scene.h"

namespace raytracer_cu {

CUDA_HOSTDEV int idx(int x, int y, int h, int w) {
  return y * w + x;
}

CUDA_GLOBAL void initScene(ScenePtr_t* devScenePtr) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(x == 0 && y == 0){
    Scene *scene = new RoomScene();
    devScenePtr[0] = scene;
  }
}

CUDA_GLOBAL void _buildScene(ScenePtr_t* devScenePtr) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(x == 0 && y == 0){
    ScenePtr_t scene = devScenePtr[0];
    scene -> buildScene();
  }
}

CUDA_GLOBAL void sceneTransform(mat3x3 transform, ScenePtr_t* aScene){
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(x == 0 && y == 0){
    Scene* scene = aScene[0];
    scene -> transform(transform);
  }
}

CUDA_GLOBAL void _addTexture(ScenePtr_t* devScenePtr, int texWidth, int texHeight, float3* texData){
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  ColorBuffer<float3>* texture = new ColorBuffer<float3>(texWidth, texHeight);
  texture -> c = texData;

  if(x == 0 && y == 0){  
    Scene* scene = devScenePtr[0];
    scene -> textures.push_back(texture);
  }
}

CUDA_GLOBAL void traceScene(uint8_t* cDevColorBuffer,
    ScenePtr_t* aScene, 
    float3 viewport_tl,
    float3 viewport_v1,
    float3 viewport_v2,
    float3 eye,
    int2 displaySize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(x > 0 && x < displaySize.x && y > 0 &&  y < displaySize.y){
    Scene* scene = aScene[0];
    
    float3 screen = viewport_tl +
                        (float(x) / displaySize.x) * viewport_v1 +
                        (float(y) / displaySize.y) * viewport_v2;

    float3 resultCol = make_float3(0.0f, 0.0f, 0.0f);
    
    int maxBounces = 3;
    Ray eyeRay(screen, screen-eye, maxBounces);
    int linIdx = idx(x, y, displaySize.x, displaySize.y);
    
    if (scene->trace(eyeRay, resultCol)) {
      cDevColorBuffer[linIdx*4+1] = uint8_t(__float2int_rn (resultCol.z*255));
      cDevColorBuffer[linIdx*4+2] = uint8_t(__float2int_rn (resultCol.y*255));
      cDevColorBuffer[linIdx*4+3] = uint8_t(__float2int_rn (resultCol.x*255));
    } else {
      cDevColorBuffer[linIdx*4+1] = 0;
      cDevColorBuffer[linIdx*4+2] = 0;
      cDevColorBuffer[linIdx*4+3] = 0;
    }
    cDevColorBuffer[linIdx*4+0] = 255;
  }
}

void checkCudaErr() {
  hipDeviceSynchronize();
  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << hipGetErrorString(err) << std::endl;
  }
}

void Renderer::addTexture(int texWidth, int texHeight, float3* texData){
  
  float3* dTexData;
  int nTexBytes = texWidth*texHeight*sizeof(float3);
  hipMalloc((void**)&dTexData, nTexBytes);
  hipMemcpy(dTexData, texData, nTexBytes, hipMemcpyHostToDevice);
  _addTexture<<<1, 1>>>(devScenePtr, texWidth, texHeight, dTexData);
}

CUDA_HOSTDEV mat3x3 getRotationMatrixX(float rotRad) {
  // Rotates around the X axis
  
  mat3x3 rot;
  rot.data[0][0] = 1.;
  rot.data[1][1] = cos(rotRad);
  rot.data[1][2] = -sin(rotRad);
  rot.data[2][1] = sin(rotRad);
  rot.data[2][2] = cos(rotRad);
  return rot;
}

CUDA_HOSTDEV mat3x3 getRotationMatrixY(float rotRad) {
  // Rotates around the Y axis

  mat3x3 rot;
  rot.data[1][1] = 1.;
  rot.data[0][0] = cos(rotRad);
  rot.data[2][0] = -sin(rotRad);
  rot.data[0][2] = sin(rotRad);
  rot.data[2][2] = cos(rotRad);
  return rot;
}

CUDA_HOSTDEV mat3x3 getRotationMatrixZ(float rotRad) {
  // Rotates around the z axis
  
  mat3x3 rot;
  rot.data[2][2] = 1.;
  rot.data[0][0] = cos(rotRad);
  rot.data[1][0] = -sin(rotRad);
  rot.data[0][1] = sin(rotRad);
  rot.data[1][1] = cos(rotRad);
  return rot;
}

void Renderer::mouseMoveInput(int x, int y){
  horizontalDisplacement = x;
  verticalDisplacement = y;
}

void Renderer::mouseWheelInput(int w){
}

void Renderer::keyboardArrowsInput(int x, int y){
  horizontalNavigation = x;
  verticalNavigation = y;
}

void Renderer::buildScene(){
  _buildScene<<<1, 1>>>(devScenePtr);
}

std::tuple<float3, float3, float3> getViewport(int2 viewport_size,
                                               float viewport_z) {
  // The viewport is an (assumed) square surface that is on the xy and the
  // center is at the origin.
  float3 vp = make_float3(viewport_size.x, viewport_size.y, viewport_z);

  // Define the top-left, top-right, bottom-lroteft points
  float3 viewport_tl = make_float3(-(vp.x / 2.0f), -(vp.y / 2.0f), vp.z);
  float3 viewport_tr = make_float3(vp.x / 2.0f, -(vp.y / 2.0f), vp.z);
  float3 viewport_bl = make_float3(-(vp.x / 2.0f), vp.y / 2.0f, vp.z);

  // Compute the directions
  float3 viewport_v1 = (viewport_tr - viewport_tl);
  float3 viewport_v2 = (viewport_bl - viewport_tl);

  return {viewport_tl, viewport_v1, viewport_v2};
}

Renderer::Renderer(uint32_t screen_width, uint32_t screen_height)
    : textures(EasyVector<ColorBuffer<float3> *, int>(12)) {

  displaySize = make_int2(screen_width, screen_height);

  textures = EasyVector<ColorBuffer<float3> *, int>(12);

  // Eye position
  float eye_z = -200.0f;
  eye = make_float3(0.0f, 0.0f, eye_z);

  // Config viewport
  auto viewport = getViewport(make_int2(256.0f, 256.0f), eye_z + 100.0f);
  std::tie(viewport_tl, viewport_v1, viewport_v2) = viewport;

  // Initialize CUDA stack
  size_t s;
  hipDeviceSetLimit(hipLimitStackSize, 1024*30);
  checkCudaErr();

  hipDeviceGetLimit(&s, hipLimitStackSize);
  checkCudaErr();
  std::cout << "Stack size: " << s << std::endl;

  // Initialize the color buffer
  cColBuffSizeBytes = displaySize.x * displaySize.y * 4;
  hipMalloc((void **)&cDevColorBuffer, cColBuffSizeBytes);
  checkCudaErr();

  // Initialize the scene
  hipMalloc((ScenePtr_t **)&devScenePtr, sizeof(ScenePtr_t));
  initScene<<<1, 1>>>(devScenePtr);
  checkCudaErr();
}

void Renderer::modelTransform() {
  mat3x3 rot1 = getRotationMatrixX(0.);
  mat3x3 rot2 = getRotationMatrixY(0.);
  mat3x3 transform = mm(rot2, rot1);
  sceneTransform<<<1, 1>>>(transform, devScenePtr);
}

void Renderer::viewTransform(){
  // Transform camera and eye

  sensitivity = 500.0f;

  mat3x3 rot1 = getRotationMatrixY(horizontalDisplacement/sensitivity);
  mat3x3 rot2 = getRotationMatrixX(-verticalDisplacement/sensitivity);
  mat3x3 transform = mm(rot2, rot1);

  viewport_tl = mm<3>(transform, viewport_tl);
  viewport_v1 = mm<3>(transform, viewport_v1);
  viewport_v2 = mm<3>(transform, viewport_v2);
  eye = mm<3>(transform, eye);

  horizontalDisplacement = 0;
  verticalDisplacement = 0;
}

void Renderer::render(uint8_t* frameBuffer) {
  modelTransform();
  viewTransform();

  // 32x32 blocks, 16x16 threads per block
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(displaySize.x / threadsPerBlock.x,
                 displaySize.y / threadsPerBlock.y);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  traceScene<<<numBlocks, threadsPerBlock>>>(cDevColorBuffer, devScenePtr,
                                         viewport_tl, viewport_v1, viewport_v2,
                                         eye, displaySize);
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  checkCudaErr();

  std::cout << "Trace kernel execution time: " << milliseconds << " ms." << std::endl;

  hipMemcpy(frameBuffer, cDevColorBuffer,
             cColBuffSizeBytes, hipMemcpyDeviceToHost);

}

Renderer::~Renderer(){
  hipFree(cDevColorBuffer);
}

} // namespace raytracer_cu
